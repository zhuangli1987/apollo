#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2017 The Apollo Authors. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/

#include "examples/component/perception.h"

#include "hip/hip_runtime.h"
#include "cybertron/common/log.h"
#include "cybertron/croutine/system/cuda_async.h"

__global__ void
VectorAdd(const float *A, const float *B, float *C, int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < numElements) {
    for (int j = 0; j < 1000; ++j) {
      C[i] = A[i] + B[i];
    }
  }
}

int TestCUDA() {
  if (hipSetDevice(0) != hipSuccess) {
    return -1;
  }
  //hipError_t err = hipSuccess;
  int numElements = 10000000;
  size_t size = numElements * sizeof(float);

  float *h_A = (float *)malloc(size);
  float *h_B = (float *)malloc(size);
  float *h_C = (float *)malloc(size);
  for (int i = 0; i < numElements; ++i) {
      h_A[i] = rand()/(float)RAND_MAX;
      h_B[i] = rand()/(float)RAND_MAX;
  }

  float *d_A = NULL;
  hipMalloc((void **)&d_A, size);
  float *d_B = NULL;
  hipMalloc((void **)&d_B, size);
  float *d_C = NULL;
  hipMalloc((void **)&d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  hipStream_t stream;
  hipStreamCreate(&stream);

  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  VectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, numElements);
  apollo::cybertron::croutine::CudaAsync(stream);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  for (int i = 0; i < numElements; ++i) {
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
        AERROR << "Result verification failed at element " <<  i;
        return -1;
    }
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);
  return 0;
}
